#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <iostream>
#include <sys/time.h>
#include <math.h>

using namespace std;

__global__ void add(float* x, float* y, float* z, int n){
    //huoqu suoyin
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    //buchang
    z[index] = x[index] + y[index];
    
}

int main(){

    //shijian
    struct timeval start, end;

    //创建cuda stream
    hipStream_t stream;
    hipStreamCreate(&stream);


    int N = 1024 * 1024 * 100;
    int nBytes = N * sizeof(float);
    float *x, *y, *z;
    float *d_x, *d_y, *d_z;

    //gpu 内存分配
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_z, nBytes);

    //cpu 内存锁定
    hipHostAlloc((void **)&x, nBytes, hipHostMallocDefault);
    hipHostAlloc((void **)&y, nBytes, hipHostMallocDefault);
    hipHostAlloc((void **)&z, nBytes, hipHostMallocDefault);

    for(int i =0; i<N; ++i){
        x[i] = 10.0;
        y[i] = 20.0;
    }

    //dingyi kernel peizhi
    dim3 blockSize(1024);
    dim3 gridSize(N / 1024);

    gettimeofday( &start, NULL );

    //使用stream 数据传输与计算同步进行
    for (int i = 0; i < N; i += N/25){

		hipMemcpyAsync(d_x, x + i, nBytes, hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(d_y, y + i, nBytes, hipMemcpyHostToDevice, stream);
 
		add<<<gridSize, blockSize, 0, stream >>>(d_x, d_y, d_z, N);
 
		hipMemcpyAsync(z + i, d_z, nBytes, hipMemcpyDeviceToHost, stream);
    }

    // 同步device 保证结果能正确访问
    hipStreamSynchronize(stream);

    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    printf("total time is %d ms\n", timeuse/1000);
    
    std::cout<<z[0]<<endl;

    //shifang neicun 
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}