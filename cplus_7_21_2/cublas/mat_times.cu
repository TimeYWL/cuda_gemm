#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>
#include "hipblas.h"

using namespace std;

int main(void){
    //shijian
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    float *A, *B, *C;
    int m, n, k;
    float alpha=1, beta=0;
    m = 4096;
    n = 4096;
    k = 4096;
    float *D_A, *D_B, *D_C;

    A = (float*)malloc(m * k * sizeof(float));
    B = (float*)malloc(k * n * sizeof(float));
    C = (float*)malloc(m * n * sizeof(float));

    hipMalloc((void **)&D_A, m * k * sizeof(float));
    hipMalloc((void **)&D_B, m * k * sizeof(float));
    hipMalloc((void **)&D_C, m * k * sizeof(float));
    
    //cublas
    hipblasHandle_t handle; 

    //赋值
    for(int i=0;i<m*k;++i){
        A[i] = 2.0;
    }
    for(int i=0;i<k*n;++i){
        B[i] = 2.0;
    }

    hipblasCreate(&handle);

    hipblasSetMatrix(m, k, sizeof(float), A, m, D_A, m);
    hipblasSetMatrix(k, n, sizeof(float), B, k, D_B, k);    
    
    hipEventRecord(start);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, D_A, m, D_B, k, &beta, D_C, m);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time, flops;
    hipEventElapsedTime(&time, start, stop);

    flops = m * n * (k * 2.0) / (time / 1000) / (1000000000);
	
	std::cout<<"Time is "<<time<<std::endl;
    std::cout<<"Flops is "<<flops<<std::endl;

    hipblasGetMatrix(m, n, sizeof(float), D_C, m, C, m);

    cout<<C[0]<<endl;

    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);

    free(A);
    free(B);
    free(C);

    hipblasDestroy(handle);

}